#include <iostream>
#include <hip/hip_runtime.h>

__global__ void add(int *a, int *b, int *c, int n) {
    int idx = threadIdx.x;   // index within the block
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    const int N = 10; // length of arrays
    int h_a[N], h_b[N], h_c[N]; // host arrays

    // initialize host arrays
    for (int i = 0; i < N; i++) {
        h_a[i] = i;
        h_b[i] = i * 2;
    }

    int *d_a, *d_b, *d_c; // device pointers
    size_t size = N * sizeof(int);

    // allocate device memory
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    // copy host arrays to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    // launch kernel with one block of N threads
    add<<<1, N>>>(d_a, d_b, d_c, N);

    // copy result back to host
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    // print result
    std::cout << "Result: ";
    for (int i = 0; i < N; i++) {
        std::cout << h_c[i] << " ";
    }
    std::cout << std::endl;

    // free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
